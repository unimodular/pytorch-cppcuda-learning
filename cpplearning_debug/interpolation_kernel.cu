#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp2)
{
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n >= feats.size(0) || f >= feats.size(2))
        return;
    // NOTE: 这点没太懂
    // point -1~1
    const scalar_t u = (points[n][0] + 1) / 2;
    const scalar_t v = (points[n][1] + 1) / 2;
    const scalar_t w = (points[n][2] + 1) / 2;

    const scalar_t a = (1 - v) * (1 - w);
    const scalar_t b = (1 - v) * w;
    const scalar_t c = v * (1 - w);
    const scalar_t d = 1 - a - b - c;
    feat_interp[n][f] = (1 - u) * (a * feats[n][0][f] +
                                   b * feats[n][1][f] +
                                   c * feats[n][2][f] +
                                   d * feats[n][3][f]) +
                        u * (a * feats[n][4][f] +
                             b * feats[n][5][f] +
                             c * feats[n][6][f] +
                             d * feats[n][7][f]);
    feat_interp2[n][f] = (2 - u) * (a * feats[n][0][f] +
                                    b * feats[n][1][f] +
                                    c * feats[n][2][f] +
                                    d * feats[n][3][f]) +
                         u * (a * feats[n][4][f] +
                              b * feats[n][5][f] +
                              c * feats[n][6][f] +
                              d * feats[n][7][f]);
}

// torch::Tensor trilinear_fw_cu(
//     torch::Tensor feats,
//     torch::Tensor points)
std::vector<torch::Tensor> trilinear_fw_cu(
    torch::Tensor feats,
    torch::Tensor points)
{
    const int N = feats.size(0), F = feats.size(2);

    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());
    torch::Tensor feat_interp2 = torch::zeros({N, F}, feats.options());

    const dim3 threads(16, 16); // 256
    const dim3 blocks((N + threads.x - 1) / threads.x, (F + threads.y - 1) / threads.y);
    AT_DISPATCH_FLOATING_TYPES(feats.scalar_type(), "trilinear_fw_cu",
                               ([&]
                                { trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
                                      feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
                                      points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
                                      feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
                                      feat_interp2.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()); }));
    return {feat_interp, feat_interp2}; // 使用 std::vector 返回多个张量;
}